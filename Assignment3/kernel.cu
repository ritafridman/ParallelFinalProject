#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include "basicFunctions.h"

hipError_t Calculate(int n, int k, double *weights, int *arr);
hipError_t Save_Array_Points_And_Weights(point_t *pointsArr, int n, double *weights, int k);
hipError_t copy_w(double *weights, int k);
hipError_t free_All(void);

point_t *gpu_pointsArray;
double *gpu_weights;
int *dev_arr;

__global__  void GpuCudaArr(point_t* gpu_pointsArray, double* gpu_weights, int* dev_arr, int k) {
	int idInsideBlock = threadIdx.x;
	int idBlock = blockIdx.x;
	int myLocation = NUM_OF_THREADS * idBlock + idInsideBlock;
	int i;
	double sum = 0;
	for (i = 0; i < k; i++) {
		sum += (gpu_pointsArray[myLocation].values[i] * gpu_weights[i]);
	}
	if (sum >= 0)
		dev_arr[myLocation] = 1;
	else
		dev_arr[myLocation] = -1;

}

//save the point array and Weights without free:
hipError_t Save_Array_Points_And_Weights(point_t *pointsArr, int n, double *weights, int k) {
	gpu_pointsArray = 0;
	gpu_weights = 0;
	dev_arr = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers:
	cudaStatus = hipMalloc((void**)&gpu_pointsArray, n * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&gpu_weights, k * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_arr, n * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(gpu_pointsArray, pointsArr, n * sizeof(point_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, " hipDeviceSynchronize returned error code %d after launching Calculate_Bigger_Than_Zero!\n", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}



//after training:
hipError_t  copy_w(double *weights, int k) {
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(gpu_weights, weights, k * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Calculate_Bigger_Than_Zero!\n", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}


hipError_t free_All(void) {
	hipError_t cudaStatus;
	//	cudaStatus = hipSetDevice(0);
	hipFree(gpu_pointsArray);
	hipFree(gpu_weights);
	hipFree(dev_arr);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching freecuda!\n", cudaStatus);
	}
	return cudaStatus;
}

hipError_t Calculate(int n, int k, double *weights, int *arr) {
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(gpu_weights, weights, k * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with thread.
	int numBlocks = (n / NUM_OF_THREADS);
	GpuCudaArr << <numBlocks, NUM_OF_THREADS >> >(gpu_pointsArray, gpu_weights, dev_arr, k);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Calculate launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Calculate_Bigger_Than_Zero!\n", cudaStatus);
		goto Error;
	}

	// Copy output from GPU buffer to host memory.
	cudaStatus = hipMemcpy(arr, dev_arr, n * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("cudaStatus failed! %s \n", hipGetErrorString(cudaStatus));
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	return cudaStatus;

}
